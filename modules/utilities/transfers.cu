#include <stdio.h>   // Library variables (e.g. FILE), library macros (e.g. stderr, stdin, stdout), library functions (e.g. fopen, fclose, fread, fwrite, fprintf, printf, fscanf, fgets, fflush)
#include <stdlib.h>  // Library variables (e.g. size_t), library macros (e.g. EXIT_FAILURE, EXIT_SUCCESS, RAND_MAX), library functions (e.g. atof, atoi, malloc, free, exit, rand)
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>

#include "../../include/config.h"
#include "../../include/utilities/init_gpu.h"
#include "../../include/utilities/transfers.h"


// Transfer real data from host to device
void real_data_register (T_real *data, size_t size)
{
    CHECK_CUDA_SUCCESS(hipHostRegister(data, size, hipHostRegisterPortable));
}
void float_data_register (float *data, size_t size)
{
    CHECK_CUDA_SUCCESS(hipHostRegister(data, size, hipHostRegisterPortable));
}
void double_data_register (double *data, size_t size)
{
    CHECK_CUDA_SUCCESS(hipHostRegister(data, size, hipHostRegisterPortable));
}

void real_data_transfers_cpu_to_gpu (T_real *data, size_t size,
                                     T_real *GPU_data)
{   
    CHECK_CUDA_SUCCESS(hipMemcpy(GPU_data, data, size, hipMemcpyHostToDevice));
}
void float_data_transfers_cpu_to_gpu (float *data, size_t size,
                                     float *GPU_data)
{   
    CHECK_CUDA_SUCCESS(hipMemcpy(GPU_data, data, size, hipMemcpyHostToDevice));
}
void double_data_transfers_cpu_to_gpu (double *data, size_t size,
                                     double *GPU_data)
{   
    CHECK_CUDA_SUCCESS(hipMemcpy(GPU_data, data, size, hipMemcpyHostToDevice));
}



// Transfer integer data from host to device
void int_data_register (int *data, size_t size)
{
    CHECK_CUDA_SUCCESS(hipHostRegister(data, size, hipHostRegisterPortable));
}


void int_data_transfers_cpu_to_gpu (int *data, size_t size,
                                    int *GPU_data)
{   
    CHECK_CUDA_SUCCESS(hipMemcpy(GPU_data, data, size, hipMemcpyHostToDevice));
}




// Transfer real data from device to host
void real_data_transfers_gpu_to_cpu (T_real *GPU_data, size_t size,
                                     T_real *data)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(data, GPU_data, size, hipMemcpyDeviceToHost));
}
void float_data_transfers_gpu_to_cpu (float *GPU_data, size_t size,
                                     float *data)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(data, GPU_data, size, hipMemcpyDeviceToHost));
}
void double_data_transfers_gpu_to_cpu (double *GPU_data, size_t size,
                                     double *data)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(data, GPU_data, size, hipMemcpyDeviceToHost));
}

void real_data_unregister (T_real *data)
{
    CHECK_CUDA_SUCCESS(hipHostUnregister(data));
}
void float_data_unregister (float *data)
{
    CHECK_CUDA_SUCCESS(hipHostUnregister(data));
}
void double_data_unregister (double *data)
{
    CHECK_CUDA_SUCCESS(hipHostUnregister(data));
}



// Transfer integer data from device to host
void int_data_transfers_gpu_to_cpu (int *GPU_data, size_t size,
                                    int *data)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(data, GPU_data, size, hipMemcpyDeviceToHost));
}

void int_data_unregister (int *data)
{
    CHECK_CUDA_SUCCESS(hipHostUnregister(data));
}